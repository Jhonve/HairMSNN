#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright (c) Meta Platforms, Inc. and affiliates.                       //
//                                                                          //
// This source code is licensed under the MIT license found in the          //
// LICENSE file in the root directory of this source tree.                  //
// ======================================================================== //

#include "hair_msnn.cuh"
#include "utils.cuh"

#include "curve_utils.cuh"
#include "disney_hair.cuh"
#include "frostbite_anisotropic.cuh"

#include "optix_common.cuh"

__device__
vec3f msnnNextPathVertex(Interaction& si, LCGRand& rng)
{
    vec2f rand1 = vec2f(lcg_randomf(rng), lcg_randomf(rng));

    float nextPdf = 1.f;
    vec3f nextBsdf(0.f);

    if (!si.isSurface) {
        vec4f rand2 = vec4f(rand1.x, rand1.y, lcg_randomf(rng), lcg_randomf(rng));
        nextBsdf = sample_disney_hair(si, rand2, &nextPdf);
    }
    else {
        si.wi_local = sample_GGX(rand1, si.alpha, si.wo_local,
            &nextPdf);
        si.wi = normalize(apply_mat(si.to_world, si.wi_local));

        nextBsdf = frostbite_GGX(si.wo_local, si.wi_local, si.color, si.alpha);
    }

    Interaction prevSi = si;

    si.hit = false;
    si.wo = -si.wi; // Direction is outward

    float wiDotN = dot(si.wi, si.n);
    bool lowerHemi = wiDotN < 0.f;
    vec3f nd = si.n;
    if (lowerHemi && !si.isSurface) {
        nd = -si.n;
        si.p = si.p + 2.f * si.hair.radius * nd;
    }

    vec3f mulFac(0.f);
    if (nextPdf == 0.f)
        mulFac = nextBsdf;
    else
        mulFac = nextBsdf / nextPdf;

    if (isnan(mulFac.x) || isnan(mulFac.y) || isnan(mulFac.z))
        mulFac = vec3f(1.f);

    RadianceRay nextRay;
    nextRay.origin = si.p + 1e-3f * nd;
    nextRay.direction = -si.wo; // On the other hand, ray direction should point correctly!
    owl::traceRay(optixLaunchParams.world, nextRay, si);

    return mulFac;
}

__device__
vec3f msnnTrainingPath(Interaction si, LCGRand& rng, vec3f& shortPathColor)
{
    vec3f beta(1.f), betaShortPath(1.f);
    vec3f final_color(0.f);
    float indirectFactor = 1.f;
    bool isPrevHair = !si.isSurface;

    final_color = directLighting(optixLaunchParams, si, rng);
    shortPathColor = final_color;

    int bounces = 1;
    for (bounces = 1; bounces < optixLaunchParams.pathV2; bounces++) {
        /* ================================================
        Next vertex in the path
        ================================================ */
        isPrevHair = !si.isSurface;

        vec3f mulFac(0.f);
        mulFac = msnnNextPathVertex(si, rng);

        // Update betas here
        beta = beta * mulFac;
        betaShortPath = betaShortPath * mulFac;

        /* ================================================
        Terminate if escaped
        ================================================ */
        if (si.hit == false) {
            break;
        }

        /* ================================================
        Direct lighting
        ================================================ */
        vec3f dl = directLighting(optixLaunchParams, si, rng);
        
        final_color += beta * dl;
        shortPathColor += indirectFactor * betaShortPath * dl;

        /* ================================================
        Russian Roulette path termination (from PBRT)
        ================================================ */
        float q = max(0.05f, 1.f - luminance(beta));
        float qShortPath = max(0.05f, 1.f - luminance(betaShortPath));
        float eps = lcg_randomf(rng);

        bool cond1 = eps < qShortPath;
        bool cond2 = bounces > optixLaunchParams.beta;

        if (cond1 || cond2) {
            betaShortPath = 0.f;
        }

        if (eps < q) {
            break;
        }

        beta = beta / (1.f - q);

        if(betaShortPath != vec3f(0.f))
            betaShortPath = betaShortPath / (1.f - qShortPath);
    }

    return final_color;
}

__device__
vec3f msnnPathTrace(Interaction si, LCGRand& rng, int v2Stop)
{
    vec3f beta(1.f);
    vec3f final_color(0.f);
    float indirectFactor = 1.f;
    bool isPrevHair = !si.isSurface;

    final_color = directLighting(optixLaunchParams, si, rng);

    int bounces = 1;
    for (bounces = 1; bounces <= v2Stop; bounces++) {
        /* ================================================
        Next vertex in the path
        ================================================ */
        isPrevHair = !si.isSurface;
        vec3f mulFac(0.f);
        mulFac = msnnNextPathVertex(si, rng);

        // Update beta with short path factor
        beta = beta * mulFac;

        /* ================================================
        Terminate if escaped
        ================================================ */
        if (si.hit == false) {
            break;
        }

        /* ================================================
        Direct lighting
        ================================================ */
        final_color += indirectFactor * beta * directLighting(optixLaunchParams, si, rng);

        /* ================================================
        Russian Roulette path termination (from PBRT)
        ================================================ */
        float q = max(0.05f, 1.f - luminance(beta));
        float eps = lcg_randomf(rng);

        bool cond1 = eps < q;
        bool cond2 = bounces > optixLaunchParams.beta;

        if (cond1 || cond2) {
            break;
        }

        beta = beta / (1.f - q);
    }

    return final_color;
}

OPTIX_RAYGEN_PROGRAM(rayGenCam)()
{
    const RayGenData& self = owl::getProgramData<RayGenData>();
    const vec2i pixelId = owl::getLaunchIndex();
    int fbOfs = pixelId.x + self.frameBufferSize.x * pixelId.y;

    // Pseudo-random number generator
    LCGRand rng = get_rng(optixLaunchParams.accumId + 10007, make_uint2(pixelId.x, pixelId.y),
        make_uint2(self.frameBufferSize.x, self.frameBufferSize.y));

    int inputCh = optixLaunchParams.mlpInputCh;
    int outputCh = optixLaunchParams.mlpOutputCh;

    int pathV1 = optixLaunchParams.pathV1 - 1;
    int pathV2 = optixLaunchParams.pathV2 - 1;

    int trOfs = 0;
    bool isTrainingPixel = false;
    RadianceRay ray;
    if (optixLaunchParams.pass == G_BUFFER) {
        trOfs = fbOfs / optixLaunchParams.everyNth;
        int trainIdx = optixLaunchParams.trainIdxs[trOfs] % optixLaunchParams.everyNth;
        isTrainingPixel = fbOfs % optixLaunchParams.everyNth == trainIdx;

        // Shoot camera ray
        vec2f pixelOffset = vec2f(lcg_randomf(rng), lcg_randomf(rng));
        const vec2f screen = (vec2f(pixelId) + pixelOffset) / vec2f(self.frameBufferSize);

        ray.origin
            = optixLaunchParams.camera.pos;
        ray.direction
            = normalize(optixLaunchParams.camera.dir_00
                + screen.u * optixLaunchParams.camera.dir_du
                + screen.v * optixLaunchParams.camera.dir_dv);
    }
    else if (optixLaunchParams.pass == TRAIN_DATA_GEN) {
        trOfs = pixelId.x + optixLaunchParams.numTrainRecordsX * pixelId.y;
        isTrainingPixel = true;

        int sceneIdx = optixLaunchParams.sceneIndices[trOfs];
        vec3f sPoint = optixLaunchParams.sampledPoints[sceneIdx];

        ray.origin
            = optixLaunchParams.camera.pos;
        ray.direction
            = normalize(sPoint - ray.origin);
    }
    
    if (optixLaunchParams.pass == G_BUFFER || optixLaunchParams.pass == TRAIN_DATA_GEN) {
        Interaction si;
        si.hit = false;
        si.wo = -1.f * ray.direction;
        si.wi = ray.direction;
        owl::traceRay(optixLaunchParams.world, ray, si);

        vec3f color(0.f), shortPathColor(0.f);
        if (isTrainingPixel) {
            if (si.hit) {
                color = msnnTrainingPath(si, rng, shortPathColor);
            }

            if (isnan(color.x) || isnan(color.y) || isnan(color.z))
                color = vec3f(0.f);

            if (isinf(color.x) || isinf(color.y) || isinf(color.z))
                color = vec3f(1e5f);

            if (isnan(shortPathColor.x) || isnan(shortPathColor.y) || isnan(shortPathColor.z))
                shortPathColor = vec3f(0.01f);

            if (isinf(shortPathColor.x) || isinf(shortPathColor.y) || isinf(shortPathColor.z))
                shortPathColor = vec3f(1e5f);

            vec3f point = si.p / optixLaunchParams.sceneScale;
            optixLaunchParams.nnTrainInput[trOfs * inputCh + 0] = point.x;
            optixLaunchParams.nnTrainInput[trOfs * inputCh + 1] = point.y;
            optixLaunchParams.nnTrainInput[trOfs * inputCh + 2] = point.z;

            optixLaunchParams.nnTrainInput[trOfs * inputCh + 3] = si.wo.x;
            optixLaunchParams.nnTrainInput[trOfs * inputCh + 4] = si.wo.y;
            optixLaunchParams.nnTrainInput[trOfs * inputCh + 5] = si.wo.z;

            optixLaunchParams.nnTrainInput[trOfs * inputCh + 6] = si.t.x;
            optixLaunchParams.nnTrainInput[trOfs * inputCh + 7] = si.t.y;
            optixLaunchParams.nnTrainInput[trOfs * inputCh + 8] = si.t.z;

            optixLaunchParams.nnTrainOutput[trOfs * outputCh + 0] = color.x - shortPathColor.x;
            optixLaunchParams.nnTrainOutput[trOfs * outputCh + 1] = color.y - shortPathColor.y;
            optixLaunchParams.nnTrainOutput[trOfs * outputCh + 2] = color.z - shortPathColor.z;

            if (!si.hit) {
                color = si.Le;
            }
        }
        else {
            if (!si.hit) {
                color = si.Le;
            }
            else if (si.hit) {
                color = msnnPathTrace(si, rng, pathV2);
            }
        }

        if (optixLaunchParams.pass == G_BUFFER) {
            vec3f point = si.p / optixLaunchParams.sceneScale;

            optixLaunchParams.nnFrameInput[fbOfs * inputCh + 0] = point.x;
            optixLaunchParams.nnFrameInput[fbOfs * inputCh + 1] = point.y;
            optixLaunchParams.nnFrameInput[fbOfs * inputCh + 2] = point.z;

            optixLaunchParams.nnFrameInput[fbOfs * inputCh + 3] = si.wo.x;
            optixLaunchParams.nnFrameInput[fbOfs * inputCh + 4] = si.wo.y;
            optixLaunchParams.nnFrameInput[fbOfs * inputCh + 5] = si.wo.z;

            optixLaunchParams.nnFrameInput[fbOfs * inputCh + 6] = si.t.x;
            optixLaunchParams.nnFrameInput[fbOfs * inputCh + 7] = si.t.y;
            optixLaunchParams.nnFrameInput[fbOfs * inputCh + 8] = si.t.z;

            GBuffer buf;
            buf.hit = si.hit;
            buf.isSurface = si.isSurface;
            buf.p = si.p;
            buf.shortPathColor = color;

            optixLaunchParams.gBuffer[fbOfs] = buf;
        }
    }
    else if (optixLaunchParams.pass == RENDER) {
        // Read GBuffer
        GBuffer gBuffer = optixLaunchParams.gBuffer[fbOfs];

        // NRC
        vec3f nnOutput(0.f);
        nnOutput.x = optixLaunchParams.nnFrameOutput[fbOfs * outputCh + 0];
        nnOutput.y = optixLaunchParams.nnFrameOutput[fbOfs * outputCh + 1];
        nnOutput.z = optixLaunchParams.nnFrameOutput[fbOfs * outputCh + 2];

        vec3f color(0.f);
        if (!gBuffer.hit || gBuffer.isSurface) {
            // Escaped or hit surface
            color = gBuffer.shortPathColor;
            nnOutput = color;
        }
        else {
            color = gBuffer.shortPathColor + nnOutput;
        }

        // Write final color
        if (optixLaunchParams.accumId > 0) {
            gBuffer.shortPathColor = gBuffer.shortPathColor + vec3f(optixLaunchParams.ptAccumBuffer[fbOfs]);
            nnOutput = nnOutput + vec3f(optixLaunchParams.nnAccumBuffer[fbOfs]);
            color = color + vec3f(optixLaunchParams.finalAccumBuffer[fbOfs]);
        }

        optixLaunchParams.ptAccumBuffer[fbOfs] = vec4f(gBuffer.shortPathColor, 1.f);
        optixLaunchParams.nnAccumBuffer[fbOfs] = vec4f(nnOutput, 1.f);
        optixLaunchParams.finalAccumBuffer[fbOfs] = vec4f(color, 1.f);

        gBuffer.shortPathColor = (1.f / (optixLaunchParams.accumId + 1)) * gBuffer.shortPathColor;
        nnOutput = (1.f / (optixLaunchParams.accumId + 1)) * nnOutput;
        color = (1.f / (optixLaunchParams.accumId + 1)) * color;

        optixLaunchParams.ptAverageBuffer[fbOfs] = vec4f(gBuffer.shortPathColor, 1.f);
        optixLaunchParams.nnAverageBuffer[fbOfs] = vec4f(nnOutput, 1.f);
        optixLaunchParams.finalAverageBuffer[fbOfs] = vec4f(color, 1.f);

        self.frameBuffer[fbOfs] = owl::make_rgba(vec3f(linear_to_srgb(color.x),
            linear_to_srgb(color.y),
            linear_to_srgb(color.z)));
    }
}